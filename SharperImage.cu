#include "hip/hip_runtime.h"
#include <opencv2/opencv.hpp>
#include <opencv2/imgproc/imgproc.hpp>
#include <opencv2/highgui/highgui.hpp>
#include <iostream>
#include <numeric>
#include <stdlib.h>


#define BLUR_SIZE 5
#define USE_2D 0

//define the storage for the blur kernel in GPU Constant Memory
__constant__ float M_d[BLUR_SIZE];

cv::Mat imageRGBA;
cv::Mat imageGrey;
cv::Mat image;
uchar4 *d_rgbaImage__;
uchar4 *d_greyImage__;

size_t numRows() { return imageRGBA.rows; }
size_t numCols() { return imageRGBA.cols; }
const long numPixels = numRows() * numCols();


//returns a pointer to an RGBA version of the input image
//and a pointer to the single channel grey-scale output
//on both the host and device
void preProcess(uchar4 **inputImage, uchar4 **greyImage,
				uchar4 **d_rgbaImage, uchar4 **d_greyImage,
				const std::string &filename) {
	//make sure the context initializes ok
	hipFree(0);
	//Read Image into an OpenCV Matrix
	image = cv::imread(filename.c_str(), CV_LOAD_IMAGE_COLOR);
	if (image.empty()) {
		std::cerr << "Couldn't open file: " << filename << std::endl;
		exit(1);
	}
	cv::cvtColor(image, imageRGBA, CV_BGR2RGBA);

	//allocate memory for the output
	imageRGBA.copyTo(imageGrey);
	//This shouldn't ever happen given the way the images are created
	//at least based upon my limited understanding of OpenCV, but better to check
	if (!imageRGBA.isContinuous() || !imageGrey.isContinuous()) {
		std::cerr << "Images aren't continuous!! Exiting." << std::endl;
		exit(1);
	}
	*inputImage = (uchar4 *)imageRGBA.ptr<unsigned char>(0);
	*greyImage  = (uchar4 *)imageGrey.ptr<unsigned char>(0);
	const size_t numPixels = numRows() * numCols();

	//allocate memory on the device for both input and output
	hipMalloc((void **) d_rgbaImage, numPixels * sizeof(uchar4));
	hipMalloc((void **) d_greyImage, numPixels * sizeof(uchar4));

	//copy input array to the GPU
	hipMemcpy(*d_rgbaImage, *inputImage, numPixels * sizeof(uchar4), hipMemcpyHostToDevice);	

	d_rgbaImage__ = *d_rgbaImage;
	d_greyImage__ = *d_greyImage;
}

void postProcess(const std::string& output_file) {
	//TODO copy the output back to the host
	hipMemcpy(imageGrey.ptr<uchar4>(0), d_greyImage__, numPixels * sizeof(uchar4), hipMemcpyDeviceToHost);	
	
	hipDeviceSynchronize();
	//change in color space required by OpenCV	
	cv::cvtColor(imageGrey, imageGrey, CV_BGR2RGBA);
	//output the image to a file
	cv::imwrite(output_file.c_str(), imageGrey);
	//display the output image (will only work if you are on the lab machines)
	cv::imshow ("Output Image", imageGrey);
	cv::waitKey(0);
	////cleanup
	hipFree(d_rgbaImage__);
	hipFree(d_greyImage__);

}

__device__ unsigned char check(int n) {return n > 255 ? 255 : (n < 0 ? 0:n);}
__device__  int indexBounds(int ndx, int maxNdx) {
   return ndx > (maxNdx - 1) ? (maxNdx - 1) : (ndx < 0 ? 0 : ndx);
}

__device__ int linearize(int c, int r, int w, int h) {
   return indexBounds(c, w) + indexBounds(r, h)*w;
}

__global__
void conv1D(const uchar4* const rgbaImage,uchar4* const greyImage,int numRows, int numCols)
{

	int pix_x = (blockIdx.x * blockDim.x) + threadIdx.x;
	int pix_y = (blockIdx.y * blockDim.y) + threadIdx.y;
   int 
	
	uchar4 blurVal = {0,0,0,1};
	for (int i = -2; i <= 2; i++) {
		cur_x = pix_x + i;
		if (cur_x >= 0 && cur_x<numCols) {
			blurVal.x += rgbaImage[pix_y * numCols + cur_x].x * M_d[i + 2];
			blurVal.y += rgbaImage[pix_y * numCols + cur_x].y * M_d[i + 2];
			blurVal.z += rgbaImage[pix_y * numCols + cur_x].z * M_d[i + 2];
		}
	}
	greyImage[pix_y * numCols + pix_x] = blurVal;
}


void your_rgba_to_greyscale(const uchar4 * const h_rgbaImage,
							uchar4 * d_rgbaImage,
							uchar4*  d_greyImage,
							size_t numRows,
							size_t numCols)
{
	float M_h[BLUR_SIZE]={0.0625, 0.25, 0.375, 0.25, 0.0625};  //change this to whatever 1D filter you are using
	hipMemcpyToSymbol(HIP_SYMBOL(M_d),M_h, BLUR_SIZE*sizeof(float)); //allocates/copy to Constant Memory on the GPU
	//temp image
	uchar4 *d_greyImageTemp;
	hipMalloc((void **)&d_greyImageTemp, sizeof(uchar4) * numRows*numCols);
	hipMemset(d_greyImageTemp, 0, numRows*numCols * sizeof(uchar4)); //make sure no memory is left laying around
	
	int threadSize=16;
	int gridSizeX=(numCols + threadSize-1)/threadSize; 
	int gridSizeY=(numRows + threadSize-1)/threadSize;
	const dim3 blockSize(threadSize, threadSize, 1);
	const dim3 gridSize(gridSizeY, gridSizeX, 1);
	for (int i=0;i<1;i++){
		//row
		conv1D<<<gridSize, blockSize>>>(d_rgbaImage,d_greyImageTemp,numRows,numCols);
		hipDeviceSynchronize();
		//col
		conv1DCol<<<gridSize, blockSize>>>(d_greyImageTemp,d_greyImage,numRows,numCols);
		hipDeviceSynchronize();

		//swap
		d_rgbaImage=d_greyImage;
	}

}

int main(int argc, char **argv) {
	hipDeviceReset();

	uchar4 *h_rgbaImage, *d_rgbaImage;
	uchar4 *h_greyImage, *d_greyImage;
	std::string input_file;
	std::string output_file;

	if (argc == 3) {
		input_file  = std::string(argv[1]);
		output_file = std::string(argv[2]);
	}
	else {
		std::cerr << "Usage: ./hw input_file output_file" << std::endl;
		exit(1);
	}

	//load the image and give us our input and output pointers
	preProcess(&h_rgbaImage, &h_greyImage, &d_rgbaImage, &d_greyImage, input_file);
	//call the students' code
	your_rgba_to_greyscale(h_rgbaImage, d_rgbaImage, d_greyImage, numRows(), numCols());
    hipDeviceSynchronize();
	hipGetLastError();
	printf("\n");
	postProcess(output_file); //prints gray image

     hipDeviceReset();
     return 0;

}

